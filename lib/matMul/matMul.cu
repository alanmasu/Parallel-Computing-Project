#include "hip/hip_runtime.h"
#include "matMul.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <cstdlib>
#include <mma.h>

using namespace nvcuda;

// Dimensione del fragment
#ifndef TILE_SIZE
    #define TILE_SIZE 16
#endif

void serialBatchedMatMul(const float *A, const float *B, float *C, int N, int blockSize){
    //int blockCount = N / blockSize;
    // Initialize the result matrix at zeros
    //memset(C, 0, N * N * sizeof(float));
    // Loop over destination blocks rows
    
}

void serialMatMul(const float *A, const float *B, float *C, int N){
    for(int r = 0; r < N; ++r){
        for(int c = 0; c < N; ++c){
            for(int k = 0; k < N; ++k){
                C[r * N + c] += A[r * N + k] * B[k * N + c];
            }
        }
    }
}

// Funzione helper per il controllo degli errori CUDA
hipError_t checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        printf("CUDA error: %s: %s\n", msg, hipGetErrorString(err));
        //exit(EXIT_FAILURE);
    }
    return err;
}

// Funzione helper per il controllo degli errori cuBLAS
void checkCublasError(hipblasStatus_t status, const char *msg) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS error: %s\n", msg);
        //exit(EXIT_FAILURE);
    }
}

// Funzione per la moltiplicazione di matrici su GPU con cuBLAS
void cublasMatMul(const float *d_A, const float *d_B, float *d_C, int N, float* milliseconds, double* TFLOPS){
    if(d_A != NULL && d_B != NULL && d_C != NULL){
        float alpha = 1.0f, beta = 0.0f;

        // Inizializzazione dell'handle cuBLAS
        hipblasHandle_t handle;
        checkCublasError(hipblasCreate(&handle), "Inizializzazione cuBLAS");

        // Misurazione del tempo
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Avvia il timer
        hipEventRecord(start, 0);

        // Esegui la moltiplicazione di matrici (C = alpha * A * B + beta * C) sulla GPU
        checkCublasError(
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N),
            "Moltiplicazione di matrici"
        );

        // Ferma il timer
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // Calcola il tempo impiegato
        if(milliseconds != NULL){
            *milliseconds = 0;
            hipEventElapsedTime(milliseconds, start, stop);
        }
        
        // Numero totale di operazioni in virgola mobile (FLOP)
        double FLOPs = 2.0 * N * N * N;

        // Calcolo dei TFLOPS
        if(milliseconds != NULL && TFLOPS != NULL){
            *TFLOPS = (FLOPs / (*milliseconds / 1000.0)) / 1e12;
        }else{
            printf("some pointers are NULL\n");
        }

        // Distruggi l'handle cuBLAS
        hipblasDestroy(handle);
    }else{
        printf("unable to perform MatMul caused by NULL pointers\n");
        if(milliseconds != NULL && TFLOPS != NULL){
            *milliseconds = -1;
            *TFLOPS = -1;
        }
    }
}


// Kernel per la moltiplicazione di matrici usando Tensor Cores e WMMA
__global__ void matrixMultiplyTensorCore(const half *a, const half *b, float *c, int M) {
    // Matrici WMMA (warped matrix multiply and accumulate)
    wmma::fragment<wmma::matrix_a, TILE_SIZE, TILE_SIZE, TILE_SIZE, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, TILE_SIZE, TILE_SIZE, TILE_SIZE, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, TILE_SIZE, TILE_SIZE, TILE_SIZE, float> c_frag;

    // Coordinate di blocco e thread
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Ogni warp calcola una tile C del risultato
    wmma::fill_fragment(c_frag, 0.0f);

    // Itera sui blocchi di K (MATRIX_SIZE / TILE_SIZE)
    for (int tileIdx = 0; tileIdx < M / TILE_SIZE; ++tileIdx) {
        // Carica una tile da A e B
        wmma::load_matrix_sync(a_frag, a + blockRow * TILE_SIZE * M + tileIdx * TILE_SIZE, M);
        wmma::load_matrix_sync(b_frag, b + tileIdx * TILE_SIZE * M + blockCol * TILE_SIZE, M);

        // Esegui la moltiplicazione delle tile
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }

    // Scrivi il risultato nella matrice C
    wmma::store_matrix_sync(c + blockRow * TILE_SIZE * M + blockCol * TILE_SIZE, c_frag, M, wmma::mem_row_major);
}


// Funzione per la moltiplicazione di matrici su GPU con Tensor Cores e WMMA
void tensorCoreMatMul(const float *h_A, const float *h_B, float *d_C, int N, float* milliseconds, double* TFLOPS) {
    // Allocazione delle matrici A e B sul device
    int device_matrix_size = N * N * sizeof(half);
    half *A, *B, *h_Ahalf, *h_Bhalf;
    
    hipError_t err1 = checkCudaError(hipMalloc((void **)&A, device_matrix_size), "Allocazione matrice A su GPU");
    hipError_t err2 = checkCudaError(hipMalloc((void **)&B, device_matrix_size), "Allocazione matrice B su GPU");
    
    //Conversione delle matrici in half
    if( A != NULL && B != NULL && err1 == hipSuccess && err2 == hipSuccess){
        for(int i = 0; i < N * N; ++i){
            h_Ahalf[i] = __float2half(h_A[i]);
            h_Bhalf[i] = __float2half(h_B[i]);
        }
        checkCudaError(hipMemcpy(A, h_Ahalf, device_matrix_size, hipMemcpyHostToDevice), "Copia matrice A sulla GPU");
        checkCudaError(hipMemcpy(B, h_Bhalf, device_matrix_size, hipMemcpyHostToDevice), "Copia matrice B sulla GPU");

    }else{
        printf("unable to perform MatMul caused by NULL pointers\n");
        if(milliseconds != NULL && TFLOPS != NULL){
            *milliseconds = -1;
            *TFLOPS = -1;
        }
        return;
    }
    // Configura la griglia e i blocchi per la computazione
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(N / TILE_SIZE, N / TILE_SIZE);

    // Misurazione del tempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Avvia il timer
    hipEventRecord(start, 0);
    
    // Esegui il kernel per la moltiplicazione di matrici con Tensor Cores e WMMA
    matrixMultiplyTensorCore<<<numBlocks, threadsPerBlock>>>(A, B, d_C, N);
    
    // Ferma il timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calcola il tempo impiegato
    if(milliseconds != NULL){
        *milliseconds = 0;
        hipEventElapsedTime(milliseconds, start, stop);
    }
    
    // Numero totale di operazioni in virgola mobile (FLOP)
    double FLOPs = 2.0 * N * N * N;

    // Calcolo dei TFLOPS
    if(milliseconds != NULL && TFLOPS != NULL){
        *TFLOPS = (FLOPs / (*milliseconds / 1000.0)) / 1e12;
    }else{
        printf("some pointers are NULL\n");
    }

    // Libera la memoria sul device
    hipFree(A);
    hipFree(B);

    // Libera la memoria sull'host
    free(h_Ahalf);
    free(h_Bhalf);

}
