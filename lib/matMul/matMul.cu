#include "hip/hip_runtime.h"
#include "matMul.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <cstdlib>
#include <mma.h>

using namespace nvcuda;

// Dimensione del fragment
#ifndef TILE_SIZE
    #define TILE_SIZE 16
#endif

void serialBatchedMatMul(const float *A, const float *B, float *C, int N, int blockSize){
    //int blockCount = N / blockSize;
    // Initialize the result matrix at zeros
    //memset(C, 0, N * N * sizeof(float));
    // Loop over destination blocks rows
    
}

void serialMatMul(const float *A, const float *B, float *C, int N){
    for(int r = 0; r < N; ++r){
        for(int c = 0; c < N; ++c){
            for(int k = 0; k < N; ++k){
                C[r * N + c] += A[r * N + k] * B[k * N + c];
            }
        }
    }
}

// Funzione helper per il controllo degli errori CUDA
hipError_t checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        printf("CUDA error: %s: %s\n", msg, hipGetErrorString(err));
        //exit(EXIT_FAILURE);
    }
    return err;
}

// Funzione helper per il controllo degli errori cuBLAS
void checkCublasError(hipblasStatus_t status, const char *msg) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS error: %s\n", msg);
        //exit(EXIT_FAILURE);
    }
}

// Funzione per la moltiplicazione di matrici su GPU con cuBLAS
void cublasMatMul(const float *d_A, const float *d_B, float *d_C, int n, float* milliseconds, double* TFLOPS){
    if(d_A != NULL && d_B != NULL && d_C != NULL){
        float alpha = 1.0f, beta = 0.0f;

        // Inizializzazione dell'handle cuBLAS
        hipblasHandle_t handle;
        checkCublasError(hipblasCreate(&handle), "Inizializzazione cuBLAS");

        // Misurazione del tempo
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Avvia il timer
        hipEventRecord(start, 0);

        // Esegui la moltiplicazione di matrici (C = alpha * A * B + beta * C) sulla GPU
        checkCublasError(
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n),
            "Moltiplicazione di matrici"
        );

        // Ferma il timer
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // Calcola il tempo impiegato
        if(milliseconds != NULL){
            *milliseconds = 0;
            hipEventElapsedTime(milliseconds, start, stop);
        }
        
        // Numero totale di operazioni in virgola mobile (FLOP)
        double FLOPs = 2.0 * n * n * n;

        // Calcolo dei TFLOPS
        if(milliseconds != NULL && TFLOPS != NULL){
            *TFLOPS = (FLOPs / (*milliseconds / 1000.0)) / 1e12;
        }else{
            printf("some pointers are NULL\n");
        }

        // Distruggi l'handle cuBLAS
        hipblasDestroy(handle);
    }else{
        printf("unable to perform MatMul caused by NULL pointers\n");
        if(milliseconds != NULL && TFLOPS != NULL){
            *milliseconds = -1;
            *TFLOPS = -1;
        }
    }
}


// Kernel per la moltiplicazione di matrici usando Tensor Cores e WMMA
#ifndef WMMA_BATCHED
__global__ void matrixMultiplyTensorCore(const half *a, const half *b, float *c, int M) {
    // Matrici WMMA (warped matrix multiply and accumulate)
    wmma::fragment<wmma::matrix_a, TILE_SIZE, TILE_SIZE, TILE_SIZE, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, TILE_SIZE, TILE_SIZE, TILE_SIZE, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, TILE_SIZE, TILE_SIZE, TILE_SIZE, float> c_frag;

    // Coordinate di blocco e thread
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Ogni warp calcola una tile C del risultato
    wmma::fill_fragment(c_frag, 0.0f);

    // Itera sui blocchi di K (MATRIX_SIZE / TILE_SIZE)
    for (int tileIdx = 0; tileIdx < M / TILE_SIZE; ++tileIdx) {
        // Carica una tile da A e B
        wmma::load_matrix_sync(a_frag, a + blockRow * TILE_SIZE * M + tileIdx * TILE_SIZE, M);
        wmma::load_matrix_sync(b_frag, b + tileIdx * TILE_SIZE * M + blockCol * TILE_SIZE, M);

        // Esegui la moltiplicazione delle tile
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }

    // Scrivi il risultato nella matrice C
    wmma::store_matrix_sync(c + blockRow * TILE_SIZE * M + blockCol * TILE_SIZE, c_frag, M, wmma::mem_row_major);
}
#else
#define WMMA_N 16
__global__ void matrixMultiplyTensorCore(const half *a, const half *b, float *d_c, int n, int bs) {
    //TODO: Implementare la gestione della shared memory

    //Creazione dei fragment
    wmma::fragment<wmma::matrix_a, WMMA_N, WMMA_N, WMMA_N, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_N, WMMA_N, WMMA_N, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_N, WMMA_N, WMMA_N, float> acc_frag;
    // wmma::fragment<wmma::accumulator, WMMA_N, WMMA_N, WMMA_N, float> c_frag;

    //Coordinate di blocco
    int bRow = blockIdx.y * blockDim.y;
    int bCol = blockIdx.x * blockDim.x;

    //Numero di blocchi
    int numBlocks = n / bs;

    //Coordinate di partenza del blocco in C
    int cStartingRow = bRow * n * bs;
    int cStartingCol = bCol * bs;
    //Moltiplico i blocchi BSxBS tra di loro
    for(int k = 0; k < numBlocks; ++k){
        //extern __shared__ float c_temp[];
        //Moltiplico all'interno dei blocchi BSxBS con i tensor cores
        //Simple batched matrix multiplication
        int aStartingCol = k * bs;
        int bStartingRow = k * bs * n;
        // printf("c[%d][%d] = a[%d][%d] * b[%d][%d]\n", cStartingRow, cStartingCol, cStartingRow, aStartingCol, bStartingRow, cStartingCol);
        // //Creo una matrice temporanea per il risultato (è una matrice BS x BS) moltiplicando i 
        // // rispettivi blocchi di matrici di dimensione WMMA_N x WMMA_N
        // extern __shared__ float c_temp[];
        for(int r = 0; r < bs/WMMA_N; ++r){
            for(int c = 0; c < bs/WMMA_N; ++c){
                //Coordinate di partenza del blocco in C
                int cCol = cStartingCol + c * WMMA_N;
                int cRow = cStartingRow + r * n * WMMA_N;
                //Carico il fragment di accumulazione
                wmma::load_matrix_sync(acc_frag, d_c + cRow + cCol, n, wmma::mem_row_major);
                for(int i = 0; i < bs/WMMA_N; ++i){
                    int aCol = aStartingCol + i * WMMA_N;
                    int bRow = bStartingRow + i * n * WMMA_N;
                    // printf("c[%d][%d] = a[%d][%d] * b[%d][%d] -> k=%d\n", cRow/n, cCol, cRow/n, aCol, bRow/n, cCol, k);

                    //Carico le matrici
                    wmma::load_matrix_sync(a_frag, a + cStartingRow + aCol, n);
                    wmma::load_matrix_sync(b_frag, b + bRow + cStartingCol, n);

                    // Moltiplico le matrici
                    wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
                }
                wmma::store_matrix_sync(d_c + cRow + cCol, acc_frag, n, wmma::mem_row_major);
            }
        }
        // __syncthreads();
        //Copio e accumulo il risultato nella matrice C
        // for(int r = 0; r < bs; ++r){
        //     for(int c = 0; c < bs; ++c){
        //         h_c[(cStartingRow + r * n) + cStartingCol + c] += c_temp[r * bs + c];
        //     }
        // }
    }
}
#endif // WMMA_BATCHED

hipError_t convertFloatToHalf(const float *A, half **B, int N){
    half* h_B = (half*)malloc(N * N * sizeof(half));
    if(B == NULL){
        printf("[ERROR]: unable to convert float to half caused by B NULL pointer\n");
        return hipErrorInvalidValue;
    }
    hipError_t err = hipMalloc((void **)B, N * N * sizeof(half));
    if(h_B != NULL && err == hipSuccess){
        for(int i = 0; i < N * N; ++i){
            h_B[i] = __float2half(A[i]);
        }
        err = hipMemcpy(*B, h_B, N * N * sizeof(half), hipMemcpyHostToDevice);
        free(h_B);
    }else{
        printf("[ERROR]: unable to allocate memory for half matrix\n");
    }
    return err;
}

// Funzione per la moltiplicazione di matrici su GPU con Tensor Cores e WMMA
#ifndef WMMA_BATCHED
void tensorCoreMatMul(const half *d_A, const half *d_B, float *d_C, int n, float* milliseconds, double* TFLOPS) {
    
    if(d_A == NULL || d_B == NULL || d_C == NULL){
        printf("[ERROR]: unable to perform MatMul caused by NULL pointers\n");
        if(milliseconds != NULL && TFLOPS != NULL){
            *milliseconds = -1;
            *TFLOPS = -1;
        }
        return;
    }

    // Configura la griglia e i blocchi per la computazione
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(n / TILE_SIZE, n / TILE_SIZE);

    // Misurazione del tempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Avvia il timer
    hipEventRecord(start, 0);
    
    // Esegui il kernel per la moltiplicazione di matrici con Tensor Cores e WMMA
    matrixMultiplyTensorCore<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, n);
    
    // Ferma il timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calcola il tempo impiegato
    if(milliseconds != NULL){
        *milliseconds = 0;
        hipEventElapsedTime(milliseconds, start, stop);
    }
    
    // Numero totale di operazioni in virgola mobile (FLOP)
    double FLOPs = 2.0 * n * n * n;

    // Calcolo dei TFLOPS
    if(milliseconds != NULL && TFLOPS != NULL){
        *TFLOPS = (FLOPs / (*milliseconds / 1000.0)) / 1e12;
    }else{
        printf("some pointers are NULL\n");
    }
}
#else
void tensorCoreMatMul(const half *d_A, const half *d_B, float *d_C, int n, int bs, float* milliseconds, double* TFLOPS) {
    
    if(d_A == NULL || d_B == NULL || d_C == NULL){
        printf("[ERROR]: unable to perform MatMul caused by NULL pointers\n");
        if(milliseconds != NULL && TFLOPS != NULL){
            *milliseconds = -1;
            *TFLOPS = -1;
        }
        return;
    }

    // Configura la griglia e i blocchi per la computazione
    dim3 threadsPerBlock(32);
    dim3 numBlocks(n / bs, n / bs);

    // Misurazione del tempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Avvia il timer
    hipEventRecord(start, 0);
    
    // Esegui il kernel per la moltiplicazione di matrici con Tensor Cores e WMMA
    matrixMultiplyTensorCore<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, n, bs);
    // matrixMultiplyTensorCore<<<numBlocks, 1>>>(d_A, d_B, d_C, n, bs);
    
    // Ferma il timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calcola il tempo impiegato
    if(milliseconds != NULL){
        *milliseconds = 0;
        hipEventElapsedTime(milliseconds, start, stop);
    }
    
    // Numero totale di operazioni in virgola mobile (FLOP)
    double FLOPs = 2.0 * n * n * n;

    // Calcolo dei TFLOPS
    if(milliseconds != NULL && TFLOPS != NULL){
        *TFLOPS = (FLOPs / (*milliseconds / 1000.0)) / 1e12;
    }else{
        printf("[ERROR]: some pointers are NULL\n");
    }
}
#endif // WMMA_BATCHED