#include "matMul.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <cstdlib>

void serialBatchedMatMul(const float *A, const float *B, float *C, int N, int blockSize){
    //int blockCount = N / blockSize;
    // Initialize the result matrix at zeros
    //memset(C, 0, N * N * sizeof(float));
    // Loop over destination blocks rows
    
}

void serialMatMul(const float *A, const float *B, float *C, int N){
    for(int r = 0; r < N; ++r){
        for(int c = 0; c < N; ++c){
            for(int k = 0; k < N; ++k){
                C[r * N + c] += A[r * N + k] * B[k * N + c];
            }
        }
    }
}

// Funzione helper per il controllo degli errori CUDA
void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        printf("CUDA error: %s: %s\n", msg, hipGetErrorString(err));
        //exit(EXIT_FAILURE);
    }
}

// Funzione helper per il controllo degli errori cuBLAS
void checkCublasError(hipblasStatus_t status, const char *msg) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS error: %s\n", msg);
        //exit(EXIT_FAILURE);
    }
}

void cublasMatMul(const float *d_A, const float *d_B, float *d_C, int N, float* milliseconds, double* TFLOPS){
    if(d_A != NULL && d_B != NULL && d_C != NULL){
        float alpha = 1.0f, beta = 0.0f;

        // Inizializzazione dell'handle cuBLAS
        hipblasHandle_t handle;
        checkCublasError(hipblasCreate(&handle), "Inizializzazione cuBLAS");

        // Misurazione del tempo
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Avvia il timer
        hipEventRecord(start, 0);

        // Esegui la moltiplicazione di matrici (C = alpha * A * B + beta * C) sulla GPU
        checkCublasError(
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N),
            "Moltiplicazione di matrici"
        );

        // Ferma il timer
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // Calcola il tempo impiegato
        if(milliseconds != NULL){
            *milliseconds = 0;
            hipEventElapsedTime(milliseconds, start, stop);
        }
        
        // Numero totale di operazioni in virgola mobile (FLOP)
        double FLOPs = 2.0 * N * N * N;

        // Calcolo dei TFLOPS
        if(milliseconds != NULL && TFLOPS != NULL){
            *TFLOPS = (FLOPs / (*milliseconds / 1000.0)) / 1e12;
        }else{
            printf("some pointers are NULL\n");
        }

        // Distruggi l'handle cuBLAS
        hipblasDestroy(handle);
    }else{
        printf("unable to perform MatMul caused by NULL pointers\n");
        if(milliseconds != NULL && TFLOPS != NULL){
            *milliseconds = -1;
            *TFLOPS = -1;
        }
    }
}