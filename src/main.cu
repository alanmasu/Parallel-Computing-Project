//Simple Matrix Multiplication whit cuBLAS

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Dimensione delle matrici
#define N 1024

// Funzione helper per il controllo degli errori CUDA
void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Funzione helper per il controllo degli errori cuBLAS
void checkCublasError(hipblasStatus_t status, const char *msg) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS error: %s\n", msg);
        exit(EXIT_FAILURE);
    }
}

int main() {
    // Punteri per le matrici sull'host
    float *h_A, *h_B, *h_C;
    
    // Punteri per le matrici sulla GPU (device)
    float *d_A, *d_B, *d_C;
    float alpha = 1.0f, beta = 0.0f;

    // Allocazione delle matrici sull'host (CPU)
    size_t matrix_size = N * N * sizeof(float);
    h_A = (float *)malloc(matrix_size);
    h_B = (float *)malloc(matrix_size);
    h_C = (float *)malloc(matrix_size);

    // Inizializza le matrici A e B sull'host
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Allocazione delle matrici sulla GPU (device)
    checkCudaError(hipMalloc((void **)&d_A, matrix_size), "Allocazione matrice A su GPU");
    checkCudaError(hipMalloc((void **)&d_B, matrix_size), "Allocazione matrice B su GPU");
    checkCudaError(hipMalloc((void **)&d_C, matrix_size), "Allocazione matrice C su GPU");

    // Inizializzazione dell'handle cuBLAS
    hipblasHandle_t handle;
    checkCublasError(hipblasCreate(&handle), "Inizializzazione cuBLAS");

    // Copia delle matrici dall'host alla GPU
    checkCudaError(hipMemcpy(d_A, h_A, matrix_size, hipMemcpyHostToDevice), "Copia matrice A sulla GPU");
    checkCudaError(hipMemcpy(d_B, h_B, matrix_size, hipMemcpyHostToDevice), "Copia matrice B sulla GPU");

    // Misurazione del tempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Avvia il timer
    hipEventRecord(start, 0);

    // Esegui la moltiplicazione di matrici (C = alpha * A * B + beta * C) sulla GPU
    checkCublasError(
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N),
        "Moltiplicazione di matrici"
    );

    // Ferma il timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calcola il tempo impiegato
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copia dei risultati dalla GPU all'host
    checkCudaError(hipMemcpy(h_C, d_C, matrix_size, hipMemcpyDeviceToHost), "Copia matrice C dall'host");

    // Numero totale di operazioni in virgola mobile (FLOP)
    double FLOPs = 2.0 * N * N * N;

    // Calcolo dei TFLOPS
    double TFLOPS = (FLOPs / (milliseconds / 1000.0)) / 1e12;

    // Stampa dei risultati
    printf("Tempo di esecuzione: %f ms\n", milliseconds);
    printf("TFLOPS: %f\n", TFLOPS);

    // Libera la memoria sulla GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Libera la memoria sull'host
    free(h_A);
    free(h_B);
    free(h_C);

    // Distruggi l'handle cuBLAS
    hipblasDestroy(handle);

    return 0;
}