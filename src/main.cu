//Simple Matrix Multiplication whit cuBLAS

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <matMul.h>
#include <ctime>



// Funzione per la stampa di matrici
void printMat(float *mat, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%f ", mat[i * cols + j]);
        }
        printf("\n");
    }
}

#ifndef TESTING
int main(int argc, char **argv) {
    // Recupero dell'ora corrente per la creazione del file di output
    time_t curr_time;
    tm * curr_tm;
    char filename[100];
    char descriptionFilename[100];
    time(&curr_time);
    curr_tm = localtime(&curr_time);
    strftime(filename, 99, "results/result-%d_%m_%Y-%H_%M_%S.csv", curr_tm);
    strftime(descriptionFilename, 99, "results/description-%d_%m_%Y-%H_%M_%S.txt", curr_tm);

    //Creazione del file di descrizione
    FILE *descriptionFile = fopen(descriptionFilename, "w");
    if(descriptionFile != NULL){
        fprintf(descriptionFile, "File di descrizione della run:\n");
        if(argc >= 1){
            fprintf(descriptionFile, "%s\n", argv[1]);
        }
        fclose(descriptionFile);
    }else{
        printf("Errore nella creazione del file di descrizione\n");
    }

    //Apertura file dei risultati
    FILE *resultFile = fopen(filename, "r");
    if(resultFile == NULL){
        resultFile = fopen(filename, "w");
        fprintf(resultFile, "Size,cuBLAS_ms,cuBLAS_TFLOPS,blockSize,ms,TFLOPS\n");
    }else{
        fclose(resultFile);
        resultFile = fopen(filename, "a");
    }

    // Puntatori per le matrici sull'host
    float *h_A, *h_B, *h_C;

    //Ciclo sulle size delle matrici
    for(int N = 16; N <= 16384; N *= 2){
        // Allocazione delle matrici sull'host (CPU)
        size_t matrix_size = N * N * sizeof(float);
        h_A = (float *)malloc(matrix_size);
        h_B = (float *)malloc(matrix_size);
        h_C = (float *)malloc(matrix_size);

        if(h_A == NULL || h_B == NULL || h_C == NULL){
            printf("Errore nell'allocazione delle matrici sull'host [size: %d]\n", N);
            if(resultFile != NULL){
                fclose(resultFile);
            }
            return 1;
        }

        // Inizializza le matrici A e B sull'host
        for (int i = 0; i < N * N; ++i) {
            h_A[i] = static_cast<float>(rand()) / RAND_MAX;
            h_B[i] = static_cast<float>(rand()) / RAND_MAX;
            // h_A[i] = i;
            // h_B[i] = i;
        }

        //Allocazione sul device (GPU)
        float *d_A, *d_B, *d_C;
        checkCudaError(hipMalloc((void **)&d_A, matrix_size), "Allocazione matrice A su GPU");
        checkCudaError(hipMalloc((void **)&d_B, matrix_size), "Allocazione matrice B su GPU");
        checkCudaError(hipMalloc((void **)&d_C, matrix_size), "Allocazione matrice C su GPU");

        // Copia delle matrici dall'host alla GPU
        if(d_A != NULL && d_B != NULL && d_C != NULL){
            checkCudaError(hipMemcpy(d_A, h_A, matrix_size, hipMemcpyHostToDevice), "Copia matrice A sulla GPU");
            checkCudaError(hipMemcpy(d_B, h_B, matrix_size, hipMemcpyHostToDevice), "Copia matrice B sulla GPU");
        }
        //Indicatori di performance
        float cublasMillis = 0;
        double cublasTFLOPS = 0;

        float myMillis = 0;
        double myTFLOPS = 0;

        ///////////////////// ALGORHITMs ///////////////////////
        /////// cuBLAS ///////
        // Moltiplicazione di matrici con cuBLAS
        cublasMatMul(d_A, d_B, d_C, N, &cublasMillis, &cublasTFLOPS); 
        // Copia dei risultati dalla GPU all'host
        checkCudaError(hipMemcpy(h_C, d_C, matrix_size, hipMemcpyDeviceToHost), "Copia matrice C dall'host");
        //Stampa delle matrici
        if(N <= 4){
            printf("Matrice A:\n");
            printMat(h_A, N, N);
            printf("Matrice B:\n");
            printMat(h_B, N, N);
            printf("Matrice C:\n");
            printMat(h_C, N, N);
        }

        // Stampa dei risultati
        printf("\n\nTempo di esecuzione [cuBLAS] [size: %d]: %f ms\n", cublasMillis, N);
        printf("TFLOPS [cuBLAS] [size: %d]: %f\n", cublasTFLOPS, N);

        /////// Custom Kernel ///////
        for(int bs = 16; bs <= 256; bs *= 2){
            // Moltiplicazione di matrici con kernel custom
            //tensorCoreMatMul(d_A, d_B, d_C, N, blockSize, &myMillis, &myTFLOPS);
            // Salva i risultati su file
            if(resultFile != NULL){
                fprintf(resultFile, "%d,%f,%f,%d,%f,%f\n", N, cublasMillis, cublasTFLOPS, bs, myMillis, myTFLOPS);
            }else{
                printf("[CSV]:\n");
                printf("%d,%f,%f,%d,%f,%f\n", N, cublasMillis, cublasTFLOPS, bs, myMillis, myTFLOPS);
                printf("[/CSV]\n");
            }
        }
        // Libera la memoria sull'host
        free(h_A);
        free(h_B);
        free(h_C);

        // Libera la memoria sulla GPU
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }
    if(resultFile != NULL){
        fclose(resultFile);
    }

    return 0;
}
#else

#define N 64

#warning "Testing mode"

int main(int argc, char **argv){
    printf("WMMA TEST: Testing mode\n");
    // // Puntatori per le matrici sull'host
    // float *h_A = NULL;
    // float *h_B = NULL;
    // float *h_C_cublas = NULL;
    // float *h_C_wmma = NULL;

    // // Allocazione delle matrici sull'host (CPU)
    // printf("[INFO]: Allocazione delle matrici sull'host\n");
    // size_t matrix_size = N * N * sizeof(float);
    // h_A = (float *)malloc(matrix_size);
    // h_B = (float *)malloc(matrix_size);
    // h_C_cublas = (float *)malloc(matrix_size);
    // h_C_wmma = (float *)malloc(matrix_size);
    // printf("[INFO]: Allocazione delle matrici sull'host completata\n");

    // // Inizializza le matrici A e B sull'host
    // if(h_A != NULL && h_B != NULL){
    //     printf("[INFO]: Inizializzazione delle matrici sull'host\n");
    //     for (int i = 0; i < N * N; ++i) {
    //         h_A[i] = i;
    //         h_B[i] = i;
    //     }
    // }else{
    //     printf("[ERR]:Errore nell'allocazione delle matrici sull'host\n");
    //     return 1;
    // }

    // //Allocazione sul device (GPU)
    // float *d_A = NULL, *d_B = NULL, *d_C = NULL;
    // hipError_t err1 = checkCudaError(hipMalloc((void **)&d_A, matrix_size), "Allocazione matrice A su GPU");
    // hipError_t err2 = checkCudaError(hipMalloc((void **)&d_B, matrix_size), "Allocazione matrice B su GPU");
    // hipError_t err3 = checkCudaError(hipMalloc((void **)&d_C, matrix_size), "Allocazione matrice C su GPU");

    // // Copia delle matrici dall'host alla GPU
    // if(err1 == hipSuccess && err2 == hipSuccess && err3 == hipSuccess){
    //     printf("\n[INFO]Copia delle matrici sulla GPU\n");
    //     checkCudaError(hipMemcpy(d_A, h_A, matrix_size, hipMemcpyHostToDevice), "Copia matrice A sulla GPU");
    //     checkCudaError(hipMemcpy(d_B, h_B, matrix_size, hipMemcpyHostToDevice), "Copia matrice B sulla GPU");
    //     printf("Matrici copiate sulla GPU\n");
    // }else{
    //     printf("[ERR]: Errore nell'allocazione delle matrici sulla GPU\n");
    //     return 2;
    // }
    // //Indicatori di performance
    // float cublasMillis = 0;
    // double cublasTFLOPS = 0;

    // float myMillis = 0;
    // double myTFLOPS = 0;

    // ///////////////////// ALGORHITMs ///////////////////////
    // /////// cuBLAS ///////
    // // Moltiplicazione di matrici con cuBLAS
    // cublasMatMul(d_A, d_B, d_C, N, &cublasMillis, &cublasTFLOPS); 
    // // Copia dei risultati dalla GPU all'host
    // checkCudaError(hipMemcpy(h_C_cublas, d_C, matrix_size, hipMemcpyDeviceToHost), "Copia matrice C dall'host");
    // //Stampa delle matrici
    // if(N <= 4){
    //     printf("Matrice A:\n");
    //     printMat(h_A, N, N);
    //     printf("Matrice B:\n");
    //     printMat(h_B, N, N);
    //     printf("Matrice C:\n");
    //     printMat(h_C_cublas, N, N);
    // }

    // // Stampa dei risultati
    // printf("\n\nTempo di esecuzione [cuBLAS] [size: %d]: %f ms\n", cublasMillis, N);
    // printf("TFLOPS [cuBLAS] [size: %d]: %f\n", cublasTFLOPS, N);
    
    // //Libero la memoria delle matrici sorgenti
    // hipFree(d_A);
    // hipFree(d_B);


    // /////// Custom Kernel ///////
    // // Moltiplicazione di matrici con kernel custom
    // tensorCoreMatMul(h_A, h_B, d_C, N, &myMillis, &myTFLOPS);
    // // Copia dei risultati dalla GPU all'host
    // checkCudaError(hipMemcpy(h_C_wmma, d_C, matrix_size, hipMemcpyDeviceToHost), "Copia matrice C dal device");
    // //Stampa delle matrici
    // if(N <= 4){
    //     printf("Matrice A:\n");
    //     printMat(h_A, N, N);
    //     printf("Matrice B:\n");
    //     printMat(h_B, N, N);
    //     printf("Matrice C:\n");
    //     printMat(h_C_wmma, N, N);
    // }

    // // Stampa dei risultati
    // printf("\n\nTempo di esecuzione [wmma] [size: %d]: %f ms\n", myMillis, N);
    // printf("TFLOPS [wmma] [size: %d]: %f\n", myTFLOPS, N);

    // //Testing dei risultati e confronto con cuBLAS
    // for(int i = 0; i < N * N; i++){
    //     if(h_C_cublas[i] != h_C_wmma[i]){
    //         printf("Errore: i risultati non coincidono\n");
    //         break;
    //     }
    // }


    // // Libera la memoria sull'host
    // free(h_A);
    // free(h_B);
    // free(h_C_cublas);
    // free(h_C_wmma);

    // // Libera la memoria sulla GPU
    // hipFree(d_C);
}

#endif